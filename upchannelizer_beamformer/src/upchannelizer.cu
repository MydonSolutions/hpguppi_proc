#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <hiprand.h>
#include <assert.h>
#include <hipblas.h>
#include <time.h>
#include <sys/time.h>
#include <iostream>
#include <string.h>
#include <math.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <>
#include "upchannelizer.h"

using namespace std;

// Perform transpose on the data and convert to floats
__global__
void data_transpose(signed char* data_in, hipComplex* data_tra, int offset, int n_pol, int n_chan, int n_win, int n_samp);

// Perform transpose on the output of the FFT
__global__
void fft_shift(hipComplex* data_in, hipComplex* data_tra, int offset, int n_pol, int n_coarse, int n_win, int n_fine);

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", 
            hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

signed char* d_data_char = NULL;
hipComplex* d_data_comp = NULL;
hipComplex* d_data_shift = NULL;
float* h_fft = NULL;

// Allocate memory to all arrays 
void init_FFT() {
	printf("Here In init_FFT()! \n");

	// Allocate memory for input data float type
	checkCuda(hipMalloc((void **)&d_data_char, (N_INPUT) * sizeof(signed char)));
	printf("Here 1st hipMalloc! \n");

	// Allocate memory for input data hipComplex type
	checkCuda(hipMalloc((void **)&d_data_comp, (N_INPUT) * sizeof(hipComplex) / 2));
	printf("Here 2nd hipMalloc! \n");

	// Allocate memory for data with FFT shift hipComplex type
	checkCuda(hipMalloc((void **)&d_data_shift, (N_INPUT) * sizeof(hipComplex) / 2));
	printf("Here 3rd hipMalloc! \n");

	checkCuda(hipHostMalloc((void **)&h_fft, (N_INPUT) * sizeof(float)));

	return;
}

// Perform transpose on the data and convert to floats
__global__
void data_transpose(signed char* data_in, hipComplex* data_tra, int offset, int n_pol, int n_chan, int n_win, int n_samp) {
	//int a = threadIdx.x; // Antenna index
	//int p = threadIdx.y; // Polarization index
	//int c = blockIdx.y;  // Coarse channel index
	//int w = blockIdx.x;  // Time window index
        //int t = blockIdx.z;  // Time sample index

	int t = threadIdx.x; // Time sample index
	int a = blockIdx.x;  // Antenna index
	int w = blockIdx.y;  // Time window index
	int c = blockIdx.z;  // Coarse channel index
        int p = 0;           // Polarization index

	int tb = 0; // Index for block of time samples to compensate max number of threads
	int TS = n_samp/MAX_THREADS; // Number of blocks of time samples to process

	// data_in_idx(p, t, w, c, a, Np, Nt, Nw, Nc)
	for(p=0; p<n_pol; p++){
		for(tb = 0; tb < TS; tb++){
		// If the input data is not float e.g. signed char, just multiply it by '1.0f' to convert it to a float
			int h_in = data_in_idx(p, t + tb*MAX_THREADS, w, (c + offset), a, n_pol, n_samp, n_win, n_chan); // data_in_idx(p, t, (f + offset), a, nt, n_chan);
			int h_tr = data_tr_idx(t + tb*MAX_THREADS, a, p, (c + offset), w, n_samp, n_pol, n_chan); // data_tr_idx(a, p, (f + offset), t, n_chan); (t, a, p, w, c, Nt, Np, Nw)

			data_tra[h_tr].x = data_in[2*h_in]*1.0f;
			data_tra[h_tr].y = data_in[2*h_in + 1]*1.0f;
		}
	}

	return;
}


// Perform FFT
void upchannelize(hipfftComplex* data_tra, int n_pol, int n_chan, int n_win, int n_samp){
        hipfftHandle plan;

	//int n[RANK] = {n_samp};

	// Setup the cuFFT plan
	if (hipfftPlan1d(&plan, n_samp, HIPFFT_C2C, BATCH(n_pol,n_chan)) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: Plan creation failed");
		return;	
	}
/*
	int n[RANK] = {MAX_THREADS};
	int TS = n_samp/MAX_THREADS; // Number of blocks of time samples to process
	// Setup the cuFFT plan	
	if (hipfftPlanMany(&plan, RANK, n, n, ISTRIDE, MAX_THREADS, n, OSTRIDE, MAX_THREADS, HIPFFT_C2C, BATCH(n_pol,n_chan,n_win)) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: Plan creation failed");
		return;	
	}

    	// Execute a complex-to-complex 1D FFT
	for(int tb = 0; tb < TS; tb++){
		if (hipfftExecC2C(plan, &data_tra[tb*MAX_THREADS], &data_tra[tb*MAX_THREADS], HIPFFT_FORWARD) != HIPFFT_SUCCESS){
			fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
			return;	
		}
	}
*/
	// Setup the cuFFT plan	
	//if (hipfftPlanMany(&plan, RANK, n, n, ISTRIDE, n_samp, n, OSTRIDE, n_samp, HIPFFT_C2C, BATCH(n_pol,n_chan)) != HIPFFT_SUCCESS){
	//	fprintf(stderr, "CUFFT error: Plan creation failed");
	//	return;	
	//}

    	// Execute a complex-to-complex 1D FFT
	int h = 0;
	for(int w = 0; w < n_win; w++){
		h = data_tr_idx(0, 0, 0, 0, w, n_samp, n_pol, n_chan);
		if (hipfftExecC2C(plan, &data_tra[h], &data_tra[h], HIPFFT_FORWARD) != HIPFFT_SUCCESS){
			fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
			return;	
		}
	}
}


// This kernel should only be used in the standalone code to test the FFT shift.
// If used in the upchannelized beamformer code, the amount of memory allocated will be too large
// The FFT shift will be performed during the beamforming computation to reduce the amount of memory allocated on the device (GPU)
// Perform transpose on the output of the FFT
__global__
void fft_shift(hipComplex* data_in, hipComplex* data_tra, int offset, int n_pol, int n_coarse, int n_win, int n_fine) {
        // 'f' is the largest dimension and is sometimes larger than 1024 which is the max number of threads
        // So 'f' should be the blockIdx.x which has the largest max value (over 2e9 elements)
        // Since 'f' has to be the fasted moving index for the hipfftExecC2C(), but 'a' needs to be the fastest moving index
        // for beamforming, then threadIdx.x can't be used for 'a' in this case, and threadIdx.y must be used instead.
        // blockIdx.x can't be a faster moving dimension than threadIdx.x
	int a = threadIdx.y; // Antenna index
	int p = threadIdx.z; // Polarization index
	int f = blockIdx.x;  // Fine channel index
	int c = blockIdx.y;  // Coarse channel index
	int w = blockIdx.z;  // Time window index

        int h_in = 0;
	int h_sh = 0;

	if(f < (n_fine/2)){
		h_in = data_fft_out_idx(f, a, p, (c + offset), w, n_fine, n_pol, n_coarse); 
		h_sh = data_fftshift_idx(a, p, (f+(n_fine/2)), (c + offset), w, n_pol, n_fine, n_coarse);

		data_tra[h_sh].x = data_in[h_in].x;
		data_tra[h_sh].y = data_in[h_in].y;
	}else if((f >= (n_fine/2)) && (f < n_fine)){
		h_in = data_fft_out_idx(f, a, p, (c + offset), w, n_fine, n_pol, n_coarse);
		h_sh = data_fftshift_idx(a, p, (f-(n_fine/2)), (c + offset), w, n_pol, n_fine, n_coarse);

		data_tra[h_sh].x = data_in[h_in].x;
		data_tra[h_sh].y = data_in[h_in].y;
	}

	return;
}

// Run FFT
float* run_FFT(signed char* data_in, int n_pol, int n_chan, int n_win, int n_samp) {

	hipError_t err_code;

        // Total number of time samples
        int nt = n_win*n_samp;

	// Transpose kernel: Specify grid and block dimensions
	//dim3 dimBlock_transpose(N_ANT, n_pol, 1);
	//dim3 dimGrid_transpose(n_samp, n_chan, n_win);

	//dim3 dimBlock_transpose(n_samp, 1, 1);
	dim3 dimBlock_transpose(MAX_THREADS, 1, 1);
	dim3 dimGrid_transpose(N_ANT, n_win, n_chan);

	// FFT shift kernel: Specify grid and block dimensions
	//dim3 dimBlock_fftshift(MAX_THREADS, 1, 1);
	//dim3 dimGrid_fftshift(N_ANT, n_win, n_chan);

	// FFT shift kernel: Specify grid and block dimensions
	dim3 dimBlock_fftshift(1, N_ANT, n_pol);
	dim3 dimGrid_fftshift(n_samp, n_chan, n_win);

	signed char* d_data_in = d_data_char;
	hipComplex* d_data_tra = d_data_comp;
	hipComplex* d_data_tra2 = d_data_shift;
	float* data_out = h_fft;

	//printf("Before hipMemcpy(HtoD) coefficients! \n");
	// Copy input data from host to device
	checkCuda(hipMemcpy(d_data_in, data_in, 2*N_ANT*n_pol*nt*n_chan*sizeof(signed char), hipMemcpyHostToDevice));

        // Perform transpose on the data and convert to floats  
        data_transpose<<<dimGrid_transpose, dimBlock_transpose>>>(d_data_in, d_data_tra, 0, n_pol, n_chan, n_win, n_samp);
        err_code = hipGetLastError();
	if (err_code != hipSuccess) {
		printf("FFT: data_transpose() kernel Failed: %s\n", hipGetErrorString(err_code));
	}

        // Upchannelize the data
        upchannelize((hipfftComplex*)d_data_tra, n_pol, n_chan, n_win, n_samp);

	// FFT shift and transpose
	fft_shift<<<dimGrid_fftshift, dimBlock_fftshift>>>(d_data_tra, d_data_tra2, 0, n_pol, n_chan, n_win, n_samp);

        // Copy input data from device to host
        checkCuda(hipMemcpy(data_out, (float *)d_data_tra2, 2*N_ANT*n_pol*nt*n_chan*sizeof(float), hipMemcpyDeviceToHost));

        return data_out;
}

// Generate simulated data
signed char* simulate_data(int n_pol, int n_chan, int nt) {
	signed char* data_sim;
	data_sim = (signed char*)calloc(N_INPUT, sizeof(signed char));

	/*
	'sim_flag' is a flag that indicates the kind of data that is simulated.
	sim_flag = 0 -> Ones
	sim_flag = 1 -> Ones placed in a particular bin (bin 3 for now)
	sim_flag = 2 -> Ones placed in a particular bin at a particular antenna (bin 3 and antenna 3 for now)
	sim_flag = 3 -> Rect placed in a particular bin at a particular antenna (bin 3 and antenna 3 for now)
	sim flag = 4 -> Simulated cosine wave
	sim flag = 5 -> Simulated complex exponential i.e. exp(j*2*pi*f0*t)
	*/
	int sim_flag = 5;
	if (sim_flag == 0) {
		for (int i = 0; i < (N_INPUT / 2); i++) {
			if(i < (N_REAL_INPUT/2)){
				data_sim[2 * i] = 1;
			}else{
				data_sim[2 * i] = 0;
			}
		}
	}
	if (sim_flag == 1) {
		// data_in_idx(p, t, w, c, a, Np, Nt, Nw, Nc)
		for (int p = 0; p < n_pol; p++) {
			for (int t = 0; t < nt; t++) {
				for (int a = 0; a < N_ANT; a++) {
					if(a < N_REAL_ANT){
						data_sim[2 * data_in_idx(p, t, 0, 2, a, n_pol, nt, 1, n_chan)] = 1;
						// data_sim[2 * data_in_idx(p, t, 0, 2, a, n_pol, nt, 1, n_chan)] = tmp;
					}
				}
			}
		}
	}
	if (sim_flag == 2) {
		// data_in_idx(p, t, w, c, a, Np, Nt, Nw, Nc)
		for (int p = 0; p < n_pol; p++) {
			for (int t = 0; t < nt; t++) {
				data_sim[2 * data_in_idx(p, t, 0, 2, 2, n_pol, nt, 1, n_chan)] = 1;
				// data_sim[2 * data_in_idx(p, t, 0, 2, 2, n_pol, nt, 1, n_chan)] = tmp;
			}
		}
	}
	if (sim_flag == 3) {
		// data_in_idx(p, t, w, c, a, Np, Nt, Nw, Nc)
		for (int p = 0; p < n_pol; p++) {
			for (int t = (1024*10); t < (nt-(1024*10)); t++) {
				data_sim[2 * data_in_idx(p, t, 0, 2, 2, n_pol, nt, 1, n_chan)] = 1;
				// data_sim[2 * data_in_idx(p, t, 0, 2, 2, n_pol, nt, 1, n_chan)] = tmp;
			}
		}
	}
	if (sim_flag == 4) {
		float freq = 1e3; // Resonant frequency

                float tmp_max = 1.0;
		float tmp_min = -1.0;

		for (int t = 0; t < nt; t++) {
			for (int f = 0; f < n_chan; f++) {
				for (int a = 0; a < N_ANT; a++) {
					if(a < N_REAL_ANT){
						// Requantize from doubles/floats to signed chars with a range from -128 to 127 
						// X polarization
						data_sim[2 * data_in_idx(0, t, 0, f, a, n_pol, nt, 1, n_chan)] = (signed char)((((cos(2 * PI * freq * t*0.000001) - tmp_min)/(tmp_max-tmp_min)) - 0.5)*256);
						//data_sim[2 * data_in_idx(0, t, 0, f, a, n_pol, nt, 1, n_chan) + 1] = 0;
						// Y polarization
						data_sim[2 * data_in_idx(1, t, 0, f, a, n_pol, nt, 1, n_chan)] = (signed char)((((2*cos(2 * PI * freq * t*0.000001) - tmp_min)/(tmp_max-tmp_min)) - 0.5)*256);
						//data_sim[2 * data_in_idx(1, t, 0, f, a, n_pol, nt, 1, n_chan) + 1] = 0;

						// X polarization
						//data_sim[2 * data_in_idx(0, t, 0, f, a, n_pol, nt, 1, n_chan)] = (cos(2 * PI * freq * t*0.000001));
						// Y polarization
						//data_sim[2 * data_in_idx(1, t, 0, f, a, n_pol, nt, 1, n_chan)] = (cos(2 * PI * freq * t*0.000001));
					}
				}
			}
		}
	}
	if (sim_flag == 5) {
		float freq = 1e3; // Resonant frequency

                float tmp_max = 1.0;
		float tmp_min = -1.0;

		for (int t = 0; t < nt; t++) {
			for (int f = 0; f < n_chan; f++) {
				for (int a = 0; a < N_ANT; a++) {
					if(a < N_REAL_ANT){
						// Requantize from doubles/floats to signed chars with a range from -128 to 127 
						// X polarization
						data_sim[2 * data_in_idx(0, t, 0, f, a, n_pol, nt, 1, n_chan)] = (signed char)((((cos(2 * PI * freq * t*0.000001) - tmp_min)/(tmp_max-tmp_min)) - 0.5)*256);
						data_sim[2 * data_in_idx(0, t, 0, f, a, n_pol, nt, 1, n_chan) + 1] = (signed char)((((sin(2 * PI * freq * t*0.000001) - tmp_min)/(tmp_max-tmp_min)) - 0.5)*256);
						//data_sim[2 * data_in_idx(0, t, 0, f, a, n_pol, nt, 1, n_chan) + 1] = 0;
						// Y polarization
						data_sim[2 * data_in_idx(1, t, 0, f, a, n_pol, nt, 1, n_chan)] = (signed char)((((2*cos(2 * PI * freq * t*0.000001) - tmp_min)/(tmp_max-tmp_min)) - 0.5)*256);
						data_sim[2 * data_in_idx(1, t, 0, f, a, n_pol, nt, 1, n_chan) + 1] = (signed char)((((sin(2 * PI * freq * t*0.000001) - tmp_min)/(tmp_max-tmp_min)) - 0.5)*256);
						//data_sim[2 * data_in_idx(1, t, 0, f, a, n_pol, nt, 1, n_chan) + 1] = 0;

						// X polarization
						//data_sim[2 * data_in_idx(0, t, 0, f, a, n_pol, nt, 1, n_chan)] = (cos(2 * PI * freq * t*0.000001));
						// Y polarization
						//data_sim[2 * data_in_idx(1, t, 0, f, a, n_pol, nt, 1, n_chan)] = (cos(2 * PI * freq * t*0.000001));
					}
				}
			}
		}
	}
	return data_sim;
}

// Free memory
void Cleanup_FFT() {
	// Free up GPU memory at the end of a program
	if (d_data_char != NULL) {
		hipFree(d_data_char);
	}
	if (d_data_comp != NULL) {
		hipFree(d_data_comp);
	}
	if (d_data_shift != NULL) {
		hipFree(d_data_shift);
	}
}


// Testing to see whether input data is as I expect it to be
float* data_test(signed char *sim_data){
	float* data_float;
	data_float = (float*)calloc(N_INPUT, sizeof(float));
	for (int ii = 0; ii < N_INPUT; ii++) { // Write up to the size of the data corresponding to 1k, 4k or 32k mode
		data_float[ii] = sim_data[ii]*1.0f;
	}
	return data_float;
}

//Comment out main() function when compiling for hpguppi
// <----Uncomment here if testing standalone code
// Test all of the kernels and functions, and write the output to
// a text file for analysis
int main() {
	printf("Here!\n");
        int n_pol = 2;
	// 1k mode
	//int n_chan = 16; 
        //int nt = 32768;
	// 4k mode
    	//int n_chan = 64;
        //int nt = 8192;
	// 32k mode
    	//int n_chan = 512;
        //int nt = 1024;

	// 5 seconds worth of processing at a time
	// 1k mode
	//int n_chan = 1; 
        //int nt = 4096*1024; // 4194304; // 2^22
	// 4k mode
    	int n_chan = 4; // 64
        int nt = 1024*1024; // 1048576; // 2^20
	// 32k mode
    	//int n_chan = 32;
        //int nt = 128*1024; // 131072; // 2^17

        int n_win = N_TIME_STI;
        int n_samp = nt/n_win;

	// Allocate memory to all arrays used by run_FFT() 
	init_FFT();

        printf("After init_FFT() \n");

	// Generate simulated data
	signed char* sim_data = simulate_data(n_pol, n_chan, nt);

        printf("After simulate_data() \n");


	// --------------------- Input data test --------------------- //
	int input_write = 1; // If input_write is set to 1, the simulated data will be written to a binary file for testing/verification

	if(input_write == 1){
		float* input_test = data_test(sim_data);

		// Write data to binary file for analysis
		char input_filename[128];

		printf("Here1!\n");

		strcpy(input_filename, "/datag/users/mruzinda/i/input_h_cufft.bin");

		printf("Here2!\n");

		FILE* input_file;

		printf("Here3!\n");

		input_file = fopen(input_filename, "w");

		printf("Here4!\n");

		fwrite(input_test, sizeof(float), N_INPUT, input_file);

		printf("Here5!\n");

		fclose(input_file);

		printf("Closed input file.\n");
	}
	// --------------------- Input data test end ------------------- //


	// Allocate memory for output array
	float* output_data;

	printf("Here6!\n");

	float time_taken = 0;
	float fft_time = 0;
	int num_runs = 1;

	// Start timing FFT computation //
	struct timespec tval_before, tval_after;

	for(int ii = 0; ii < num_runs; ii++){
		// Start timing beamformer computation //
		clock_gettime(CLOCK_MONOTONIC, &tval_before);

		// Run FFT 
                // Things to keep in mind about FFT output:
		// - FFT shift required after FFT
		// - Output may need to be divided number of FFT points
                output_data = run_FFT(sim_data, n_pol, n_chan, n_win, n_samp);

		// Stop timing FFT computation //
		clock_gettime(CLOCK_MONOTONIC, &tval_after);
		time_taken = (float)(tval_after.tv_sec - tval_before.tv_sec); //*1e6; // Time in seconds since epoch
		time_taken = time_taken + (float)(tval_after.tv_nsec - tval_before.tv_nsec)*1e-9; // Time in nanoseconds since 'tv_sec - start and end'
		fft_time += time_taken;
	}
	printf("Average FFT processing time: %f s\n", fft_time/num_runs);

	printf("Here7, FFT output: %f \n", output_data[0]);
	
	// Write data to binary file for analysis
	char output_filename[128];

	printf("Here8!\n");

	strcpy(output_filename, "/datag/users/mruzinda/o/output_d_cufft.bin");

	printf("Here9!\n");

	FILE* output_file;

	printf("Here10!\n");

	output_file = fopen(output_filename, "wb");

	printf("Here11!\n");

	fwrite(output_data, sizeof(float), (N_INPUT*n_pol*n_chan*nt)/(N_POL*N_FREQ*N_TIME), output_file);

	printf("Here12!\n");

	fclose(output_file);

	printf("Closed output file.\n");

	//free(sim_data);
	//free(output_data);	

	printf("Freed output array and unregistered arrays in pinned memory.\n");

	// Free up device memory
	//hipHostFree(h_data);
	//hipHostFree(h_coeff);
	Cleanup_FFT();

	printf("Here11!\n");

	return 0;
}
// <----Uncomment here if testing standalone code
