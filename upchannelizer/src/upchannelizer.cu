#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <hiprand.h>
#include <assert.h>
#include <hipblas.h>
#include <time.h>
#include <sys/time.h>
#include <iostream>
#include <string.h>
#include <math.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <>
#include "upchannelizer.h"

using namespace std;

// Perform transpose on the data and convert to floats
__global__
void data_transpose(signed char* data_in, hipComplex* data_tra, int offset, int n_pol, int n_chan, int n_win, int n_samp);

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", 
            hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

signed char* d_data_char = NULL;
hipComplex* d_data_comp = NULL;
float* h_fft = NULL;

// Allocate memory to all arrays 
void init_FFT() {
	printf("Here In init_FFT()! \n");

	// Allocate memory for input data float type
	checkCuda(hipMalloc((void **)&d_data_char, (N_INPUT) * sizeof(signed char)));
	printf("Here 1st hipMalloc! \n");

	// Allocate memory for input data hipComplex type
	checkCuda(hipMalloc((void **)&d_data_comp, (N_INPUT) * sizeof(hipComplex) / 2));
	printf("Here 2nd hipMalloc! \n");

	checkCuda(hipHostMalloc((void **)&h_fft, (N_INPUT) * sizeof(float)));

	return;
}

// Perform transpose on the data and convert to floats
__global__
void data_transpose(signed char* data_in, hipComplex* data_tra, int offset, int n_pol, int n_chan, int n_win, int n_samp) {
	int a = threadIdx.x; // Antenna index
	int p = threadIdx.y; // Polarization index
	int c = blockIdx.y;  // Coarse channel index
	int w = blockIdx.x;  // Time window index
        int t = blockIdx.z;  // Time sample index

	// If the input data is not float e.g. signed char, just multiply it by '1.0f' to convert it to a float
	int h_in = data_in_idx(p, w, t, (c + offset), a, n_pol, n_win, n_samp, n_chan); // data_in_idx(p, t, (f + offset), a, nt, n_chan);
	int h_tr = data_tr_idx(a, p, w, (c + offset), t, n_pol, n_win, n_chan); // data_tr_idx(a, p, (f + offset), t, n_chan);

	data_tra[h_tr].x = data_in[2*h_in]*1.0f;
	data_tra[h_tr].y = data_in[2*h_in + 1]*1.0f;
	

	return;
}


// Perform FFT
void upchannelize(hipComplex* data_tra, int n_pol, int n_chan, int n_win, int n_samp){
        hipfftHandle plan;

        // Number of branches to perform FFT on
        int n_branches = N_ANT*n_pol*n_chan*n_win;

	// Setup the cuFFT plan
    	hipfftPlan1d(&plan, n_samp, HIPFFT_C2C, n_branches);
    	
    	// Execute a complex-to-complex 1D FFT
    	hipfftExecC2C(plan, (hipfftComplex *)data_tra, (hipfftComplex *)data_tra, HIPFFT_FORWARD);
}

// Run FFT
float* run_FFT(signed char* data_in, int n_pol, int n_chan, int n_win, int n_samp) {

	hipError_t err_code;

        // Total number of time samples
        int nt = n_win*n_samp;

	// Transpose kernel: Specify grid and block dimensions
	dim3 dimBlock_transpose(N_ANT, n_pol, 1);
	dim3 dimGrid_transpose(n_samp, n_chan, n_win);

	signed char* d_data_in = d_data_char;
	hipComplex* d_data_tra = d_data_comp;
	float* data_out = h_fft;

	//printf("Before hipMemcpy(HtoD) coefficients! \n");
	// Copy input data from host to device
	checkCuda(hipMemcpy(d_data_in, data_in, 2*N_ANT*n_pol*nt*n_chan*sizeof(signed char), hipMemcpyHostToDevice));

        // Perform transpose on the data and convert to floats  
        data_transpose<<<dimGrid_transpose, dimBlock_transpose>>>(d_data_in, d_data_tra, 0, n_pol, n_chan, n_win, n_samp);
        err_code = hipGetLastError();
	if (err_code != hipSuccess) {
		printf("FFT: data_transpose() kernel Failed: %s\n", hipGetErrorString(err_code));
	}

        // Upchannelize the data
        upchannelize(d_data_tra, n_pol, n_chan, n_win, n_samp);

        // Copy input data from device to host
        checkCuda(hipMemcpy(data_out, (float *)d_data_tra, 2*N_ANT*n_pol*nt*n_chan*sizeof(float), hipMemcpyDeviceToHost));

        return data_out;
}

// Generate simulated data
signed char* simulate_data(int n_pol, int n_chan, int nt) {
	signed char* data_sim;
	data_sim = (signed char*)calloc(N_INPUT, sizeof(signed char));

	/*
	'sim_flag' is a flag that indicates the kind of data that is simulated.
	sim_flag = 0 -> Ones
	sim_flag = 1 -> Repeating sequence of 1 to 64
	sim_flag = 2 -> Sequence of 1 to 64 placed in a particular bin (bin 6 for now)
	sim flag = 3 -> Simulated sine wave
	*/
	int sim_flag = 3;
	if (sim_flag == 0) {
		for (int i = 0; i < (N_INPUT / 2); i++) {
			if(i < (N_REAL_INPUT/2)){
				data_sim[2 * i] = 1;
			}else{
				data_sim[2 * i] = 0;
			}
		}
	}
	if (sim_flag == 1) {
		int tmp = 0;
		for (int p = 0; p < n_pol; p++) {
			for (int t = 0; t < nt; t++) {
				for (int f = 0; f < n_chan; f++) {
					for (int a = 0; a < N_ANT; a++) {
						if (tmp >= N_ANT) {
							tmp = 0;
						}
						tmp = (tmp + 1) % (N_ANT+1);
						if(a < N_REAL_ANT){
							data_sim[2 * data_in_idx(p, 0, t, f, a, n_pol, 1, nt, n_chan)] = tmp;
						}else{
							data_sim[2 * data_in_idx(p, 0, t, f, a, n_pol, 1, nt, n_chan)] = 0;
						}
					}
				}
			}
		}
	}
	if (sim_flag == 2) {
		int tmp = 0;
		for (int p = 0; p < n_pol; p++) {
			for (int t = 0; t < nt; t++) {
				for (int a = 0; a < N_ANT; a++) {
					if (tmp >= N_ANT) {
						tmp = 0;
					}
					tmp = (tmp + 1) % (N_ANT+1);
					if(a < N_REAL_ANT){
						data_sim[2 * data_in_idx(p, 0, t, 5, a, n_pol, 1, nt, n_chan)] = tmp;
						data_sim[2 * data_in_idx(p, 0, t, 2, a, n_pol, 1, nt, n_chan)] = tmp;
					}else{
						data_sim[2 * data_in_idx(p, 0, t, 5, a, n_pol, 1, nt, n_chan)] = 0;
						data_sim[2 * data_in_idx(p, 0, t, 2, a, n_pol, 1, nt, n_chan)] = 0;
					}
				}
			}
		}
	}
	if (sim_flag == 3) {
		float freq = 1e3; // Resonant frequency

                float tmp_max = 1.0;
		float tmp_min = -1.0;

		for (int t = 0; t < nt; t++) {
			for (int f = 0; f < n_chan; f++) {
				for (int a = 0; a < N_ANT; a++) {
					if(a < N_REAL_ANT){
						// Requantize from doubles/floats to signed chars with a range from -128 to 127
						// X polarization
						data_sim[2 * data_in_idx(0, 0, t, f, a, n_pol, 1, nt, n_chan)] = (signed char)((((cos(2 * PI * freq * t*0.001) - tmp_min)/(tmp_max-tmp_min)) - 0.5)*256);
						data_sim[2 * data_in_idx(0, 0, t, f, a, n_pol, 1, nt, n_chan) + 1] = 0;
						// Y polarization
						data_sim[2 * data_in_idx(1, 0, t, f, a, n_pol, 1, nt, n_chan)] = (signed char)((((2*cos(2 * PI * freq * t*0.001) - tmp_min)/(tmp_max-tmp_min)) - 0.5)*256);
						data_sim[2 * data_in_idx(1, 0, t, f, a, n_pol, 1, nt, n_chan) + 1] = 0;
					}else{
						// X polarization
						data_sim[2 * data_in_idx(0, 0, t, f, a, n_pol, 1, nt, n_chan)] = 0;
						data_sim[2 * data_in_idx(0, 0, t, f, a, n_pol, 1, nt, n_chan) + 1] = 0;
						// Y polarization
						data_sim[2 * data_in_idx(1, 0, t, f, a, n_pol, 1, nt, n_chan)] = 0;
						data_sim[2 * data_in_idx(1, 0, t, f, a, n_pol, 1, nt, n_chan) + 1] = 0; // Make this negative if a different polarization is tested
					}
				}
			}
		}
	}
	return data_sim;
}

// Free memory
void Cleanup_FFT() {
	// Free up GPU memory at the end of a program
	if (d_data_char != NULL) {
		hipFree(d_data_char);
	}
	if (d_data_comp != NULL) {
		hipFree(d_data_comp);
	}
}

//Comment out main() function when compiling for hpguppi
// <----Uncomment here if testing standalone code
// Test all of the kernels and functions, and write the output to
// a text file for analysis
int main() {
	printf("Here!\n");
        int n_pol = 2;
	// 1k mode
	//int n_chan = 16; 
        //int nt = 32768;
	// 4k mode
    	int n_chan = 64;
        int nt = 8192;
	// 32k mode
    	//int n_chan = 512;
        //int nt = 1024;

        int n_win = N_TIME_STI;
        int n_samp = nt/n_win;

	// Allocate memory to all arrays used by run_FFT() 
	init_FFT();

        printf("After init_FFT() \n");

	// Generate simulated data
	signed char* sim_data = simulate_data(n_pol, n_chan, nt);

        printf("After simulate_data() \n");

	// Allocate memory for output array
	float* output_data;

	printf("Here5!\n");

	float time_taken = 0;
	float fft_time = 0;
	int num_runs = 10;

	// Start timing beamformer computation //
	struct timespec tval_before, tval_after;

	for(int ii = 0; ii < num_runs; ii++){
		// Start timing beamformer computation //
		clock_gettime(CLOCK_MONOTONIC, &tval_before);

		// Run beamformer 
                output_data = run_FFT(sim_data, n_pol, n_chan, n_win, n_samp);
		//output_data = run_beamformer(sim_data, sim_coefficients, n_chan, nt);
		//run_beamformer(h_data, h_coeff, output_data);

		// Stop timing beamforming computation //
		clock_gettime(CLOCK_MONOTONIC, &tval_after);
		time_taken = (float)(tval_after.tv_sec - tval_before.tv_sec); //*1e6; // Time in seconds since epoch
		time_taken = time_taken + (float)(tval_after.tv_nsec - tval_before.tv_nsec)*1e-9; // Time in nanoseconds since 'tv_sec - start and end'
		fft_time += time_taken;
		//printf("Time taken: %f s\n", time_taken);
	}
	printf("Average FFT processing time: %f s\n", fft_time/num_runs);

	printf("Here6, FFT output: %f \n", output_data[0]);
	
	// Write data to text file for analysis
	char output_filename[128];

	printf("Here7!\n");

	strcpy(output_filename, "output_d_cufft.txt");

	printf("Here8!\n");

	FILE* output_file;

	printf("Here9!\n");

	output_file = fopen(output_filename, "w");

	printf("Here10!\n");

	for (int ii = 0; ii < ((N_INPUT*n_pol*n_chan*nt)/(N_POL*N_FREQ*N_TIME)); ii++) { // Write up to the size of the data corresponding to 1k, 4k or 32k mode
		//fprintf(output_file, "%c\n", output_data[ii]);
		fprintf(output_file, "%g\n", output_data[ii]);
	}

	printf("Here11!\n");

	fclose(output_file);

	printf("Closed output file.\n");

	//free(sim_data);
	printf("After freeing coefficients.\n");
	//free(output_data);	

	printf("Freed output array and unregistered arrays in pinned memory.\n");

	// Free up device memory
	//hipHostFree(h_data);
	//hipHostFree(h_coeff);
	Cleanup_FFT();

	printf("Here11!\n");

	return 0;
}
// <----Uncomment here if testing standalone code
